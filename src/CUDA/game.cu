
#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int pointCalculation(const int *current, const int xPosition, const int yPosition, const int gameWidth, const int gameHeight) {

    int x = xPosition;
    int x_less = (x - 1) % gameWidth;
    int x_more = (x + 1) % gameWidth;

    int y = yPosition;
    int y_less = (y - 1) % gameHeight;
    int y_more = (y + 1) % gameHeight;


    int total = current[gameWidth * y_less + x_less] +      current[gameWidth * y_less + x] + current[gameWidth * y_less + x_more] +
                current[gameWidth * y      + x_less] + 10 * current[gameWidth * y      + x] + current[gameWidth * y      + x_more] +
                current[gameWidth * y_more + x_less] +      current[gameWidth * y_more + x] + current[gameWidth * y_more + x_more];
    
    return total;
    
}

__device__ void kernel(const int *current, int *next, const int xPosition, const int yPosition, const int gameWidth, const int gameHeight) {

    int total = pointCalculation(current, xPosition, yPosition, gameWidth, gameHeight);
    if (total == 12 || total == 13 || total == 3) {
        next[gameWidth * yPosition + xPosition] = 1;
    } else {
        next[gameWidth * yPosition + xPosition] = 0;
    }

}

__global__ void step(const int *current, int *next, const int gameWidth, const int gameHeight) {
    
    for (int xPosition = 0; xPosition < gameWidth; xPosition++) {
        for (int yPosition = blockIdx.x * blockDim.x + threadIdx.x; yPosition < gameHeight; yPosition += blockDim.x * gridDim.x) {
            kernel(current, next, xPosition, yPosition, gameWidth, gameHeight);
        }
    }


}


__host__ void initializeBoards(int *hostBoard, int* deviceBoard1, int* deviceBoard2, const int gameWidth, const int gameHeight) {
   
}

__host__ void cleanup(int *hostBoard, int* deviceBoard1, int* deviceBoard2) {
    
    free(hostBoard);
    hipFree(deviceBoard1);
    hipFree(deviceBoard2);
}


__host__ void showBoard(const int* hostBoard, const int gameWidth, const int gameHeight) {
    
    for (int row = 0; row < gameWidth; row++) // loop over the whole world
    {
        for (int col = 0; col < gameHeight; col++)
        {           
            if (hostBoard[col * gameWidth + row] == 1)
            {
                fprintf(stdout, "*");
            }
            else {
                fprintf(stdout, " ");
            }
        }
        fprintf(stdout, "\n");
    }
        fprintf(stdout, "-----------------\n");
        fprintf(stdout, "-----------------\n");
    
    
}


__host__ void run(const int gameWidth, const int gameHeight, const int iterations, const bool graphics) {

    // Setting the number of threads
    int number_of_blocks = 10;
    int threads_per_block = 10;
	
    // Allocating the CUDA status
    hipError_t cudaStatus;

    // Setting the CUDA device
    cudaStatus = hipSetDevice(0);

    // Checking if that worked
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "cudaSetDevice0 failed!\n");
        return;
    }

    int *hostBoard = nullptr;
    int *deviceBoard1 = nullptr;
    int *deviceBoard2 = nullptr;


    hostBoard = (int*)malloc(gameWidth * gameHeight * sizeof(int));

    for (int i = 0; i < gameWidth * gameHeight; i++) {
        hostBoard[i] = rand()%2;
    }

    hostBoard[gameWidth * 3 + 20] = 1;
    hostBoard[gameWidth * 4 + 20] = 1;
    hostBoard[gameWidth * 3 + 21] = 1;


    hipMalloc((void**)&deviceBoard1, gameWidth * gameHeight * sizeof(int)); 
    hipMalloc((void**)&deviceBoard2, gameWidth * gameHeight * sizeof(int)); 

    hipMemcpy(deviceBoard1, hostBoard, gameWidth*gameHeight*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(deviceBoard2, hostBoard, gameWidth*gameHeight*sizeof(int), hipMemcpyHostToDevice);


    if (hostBoard == nullptr) {
        fprintf(stderr, "hostboard is null");
    }
    else {
        showBoard(hostBoard, gameWidth, gameHeight);
    }


    if (deviceBoard1 == nullptr) {
        fprintf(stderr, "deviceBoard1 is null");
    }

    if (deviceBoard2 == nullptr) {
        fprintf(stderr, "deviceBoard1 is null");
    }


    for (int i = 0; i < iterations ; i++)
    {
        step<<<number_of_blocks, threads_per_block>>>(deviceBoard1, deviceBoard2, gameWidth, gameHeight);
        i++;
        if (graphics) {
            hipMemcpy(hostBoard, deviceBoard2, gameWidth*gameHeight*sizeof(*deviceBoard2), hipMemcpyDeviceToHost);
            showBoard(hostBoard, gameWidth, gameHeight);
        }

        if (i < iterations) {
            step<<<number_of_blocks, threads_per_block>>>(deviceBoard2, deviceBoard1, gameWidth, gameHeight);
            i++;
            if (graphics) {
                hipMemcpy(hostBoard, deviceBoard1, gameWidth*gameHeight*sizeof(*deviceBoard1), hipMemcpyDeviceToHost);
                showBoard(hostBoard, gameWidth, gameHeight);
            }
        }
    }
    
    cleanup(hostBoard, deviceBoard1, deviceBoard2);
}


int main(int argc, char *argv[]) {
    
    if (argc != 6) {
        printf("Input on this form:\n");
        printf("./game gameWidth gameHeight rand_seed timesteps graphics n_threads\n");
        printf("gameWidth is the width of the gameworld.\n");
        printf("gameheight is the height if the gameworld.\n");
        printf("rand_seed is the random seed to be used.\n");
        printf("timesteps is the amount of timesteps to be taken. ");
        printf("(use any amount of timesteps less than or equal to 0 for (nearly)infinite amount of timesteps)\n");
        printf("graphics is if you want to display the world graphically or not.\n");
        return -1;
    }


    const int gameWidth = std::atoi(argv[1]); // get input values;
    const int gameHeight = std::atoi(argv[2]);
    std::srand(std::atoi(argv[3]));
    const int timesteps = std::atoi(argv[4]);
    const bool graphics = std::atoi(argv[5]) == 1;

    
    
    run(gameWidth, gameHeight, timesteps, graphics);
    
}
